#include "hip/hip_runtime.h"

#include <iostream>
#include <stdexcept>
#include <string>

#include "grid.h"
#include "hydrostatic.h"
#include "reductions.h"
#include "star.h"
#include "utils.h"
#include "timing.h"
#include "dustdynamics.h"
#include "icevapour.h"

__global__ void setup_hydrostatic_maxtrix_device(double GM, GridRef g, 
                                                 FieldConstRef<double> cs2, FieldRef<double> out) {

    int j = threadIdx.x + blockIdx.x*blockDim.x ;
    int i = threadIdx.y + blockIdx.y*blockDim.y ;

    double val ;
    if (i < g.NR + 2*g.Nghost && j <  g.Nphi + 2*g.Nghost) {
        if (j > g.Nghost) {
            if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
                double Rm = sqrt(g.Rc(i)*g.Rc(i) + g.Zc(i,j-1)*g.Zc(i,j-1)) ;
                double Rp = sqrt(g.Rc(i)*g.Rc(i) + g.Zc(i,j)*g.Zc(i,j)) ;

                double av_cs = 0.5 * (1/cs2[cs2.index(i,j-1)] + 1/cs2[cs2.index(i,j)]) ;
                val = exp(GM * av_cs * (1/Rp - 1/Rm)) ;
            }
        } else {
            val = 1 ;
        }

        // Prevent negative densities
        out[out.index(i,j)] = max(val, 0.0) ;
    }
}

/* setup_hydrostatic_maxtrix
 * 
 * Setups up a matrix of the recurrance relation linking the density between 
 * cells. I.e. the solution of:
 *    (log P_+ - log P_0) / dz = - a_grav / cs^2
 * where 
 *    a_grav = (GM * z) / (R^2 + z^2)^(3/2).
 *  
 */
void setup_hydrostatic_matrix(const Star& star, const Grid& g, const Field<double>& cs2, 
                               Field<double>& rho) {
    dim3 threads(32, 32, 1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32, (g.NR+2*g.Nghost+31)/32 ) ;

    setup_hydrostatic_maxtrix_device<<<blocks, threads>>>(star.GM, g, cs2, rho) ;
    check_CUDA_errors("setup_hydrostatic_maxtrix") ;
}

__global__ void convert_pressure_to_density_device(GridRef g, FieldConstRef<double> cs2, 
                                                   FieldRef<double> rho) {
                                            
    int j = threadIdx.x + blockIdx.x*blockDim.x ;
    int i = threadIdx.y + blockIdx.y*blockDim.y ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) 
        rho[rho.index(i, j)] /= cs2[cs2.index(i, j)] ;
}


void convert_pressure_to_density(const Grid& g, const Field<double>& cs2,
                                 Field<double>& rho) {

    dim3 threads(32, 32, 1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32, (g.NR+2*g.Nghost+31)/32 ) ;

    convert_pressure_to_density_device<<<blocks, threads>>>(g, cs2, rho) ;
    check_CUDA_errors("convert_pressure_to_density_device") ;
}


__global__ void normalize_density_device(GridRef g, FieldRef<double> rho, 
                                         const double* Sigma, const double *norm) {

    int j = threadIdx.x + blockIdx.x*blockDim.x ;
    int i = threadIdx.y + blockIdx.y*blockDim.y ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        double Area = 0.5*(g.Re(i+1)*g.Re(i+1) - g.Re(i)*g.Re(i)) ;  
        rho(i, j) *= 0.5 * Sigma[i] * Area / norm[i] ; 
        rho(i, j) = max(rho(i, j), 1e-200);
    }
    __syncthreads() ;

    // Finally set the density in the ghost cell.
    if (i < g.NR + 2*g.Nghost && j < g.Nghost)
        rho(i, j) = rho(i, 2*g.Nghost-j-1) ;
}

void normalize_density(const Grid& g, Field<double>& rho, 
                       const CudaArray<double>& Sigma, const CudaArray<double>& norm) {
  
    dim3 threads(32, 32, 1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32, (g.NR+2*g.Nghost+31)/32 ) ;

    normalize_density_device<<<blocks, threads>>>(g, rho, Sigma.get(), norm.get()) ;
    check_CUDA_errors("normalize_density") ;
}


void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<double>& rho, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma) {

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;    
}

__global__ void _rho_from_wg(GridRef g, FieldRef<double> rho, FieldRef<Prims> w_g) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            rho(i,j) = w_g(i,j).rho;
        }
    }

}
__global__ void _wg_from_rho(GridRef g, FieldRef<double> rho, FieldRef<Prims> w_g, double gasfloor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            w_g(i,j).rho = rho(i,j) + gasfloor;
        }
    }

}

__global__ void _wg_from_rho(GridRef g, FieldRef<double> rho, FieldRef<Prims> w_g, double gasfloor, MoleculeRef mol) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            mol.rho(i,j).vap *= (rho(i,j) + gasfloor) / w_g(i,j).rho;
            w_g(i,j).rho = rho(i,j) + gasfloor;
        }
    }

}

// __global__ void _check_dust(GridRef g, FieldRef<double> rho, FieldRef<Quants> w_g, Field3DRef<Quants> q_d) {

//     int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
//     int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
//     int istride = gridDim.x * blockDim.x ;
//     int jstride = gridDim.y * blockDim.y ;

//     for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
//         for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
//             if (w_g(i,j).rho/rho(i,j) > 100. || w_g(i,j).rho/rho(i,j) < 0.01) {
//                 for (int k=0; k<q_d.Nd; k++) {
//                     q_d(i,j,k).rho = 1.e-40;
//                     q_d(i,j,k).mom_R = 0.;
//                     q_d(i,j,k).amom_phi = 0.;
//                     q_d(i,j,k).mom_Z = 0.;
//                 }
//             }
//         }
//     }

// }

void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;

    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor);  
}

void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, Field3D<Prims>& q_d, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;

    // _check_dust<<<blocks,threads>>>(g, rho, w_g, q_d);

    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor);  
}

void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, Molecule& mol, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;

    // _check_dust<<<blocks,threads>>>(g, rho, w_g, q_d);

    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor, mol);  
}