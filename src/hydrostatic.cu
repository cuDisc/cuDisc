#include "hip/hip_runtime.h"

#include <iostream>
#include <stdexcept>
#include <string>

#include "grid.h"
#include "hydrostatic.h"
#include "reductions.h"
#include "star.h"
#include "utils.h"
#include "timing.h"
#include "dustdynamics.h"
#include "icevapour.h"

__global__ void setup_hydrostatic_maxtrix_device(double GM, GridRef g, 
                                                 FieldConstRef<double> cs2, FieldRef<double> out) {

    int j = threadIdx.x + blockIdx.x*blockDim.x ;
    int i = threadIdx.y + blockIdx.y*blockDim.y ;

    double val ;
    if (i < g.NR + 2*g.Nghost && j <  g.Nphi + 2*g.Nghost) {
        if (j > g.Nghost) {
            if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
                double Rm = sqrt(g.Rc(i)*g.Rc(i) + g.Zc(i,j-1)*g.Zc(i,j-1)) ;
                double Rp = sqrt(g.Rc(i)*g.Rc(i) + g.Zc(i,j)*g.Zc(i,j)) ;

                double av_cs = 0.5 * (1/cs2[cs2.index(i,j-1)] + 1/cs2[cs2.index(i,j)]) ;
                val = exp(GM * av_cs * (1/Rp - 1/Rm)) ;
            }
        } else {
            val = 1 ;
        }

        // Prevent negative densities
        out[out.index(i,j)] = max(val, 0.0) ;
    }
}

/* setup_hydrostatic_maxtrix
 * 
 * Setups up a matrix of the recurrance relation linking the density between 
 * cells. I.e. the solution of:
 *    (log P_+ - log P_0) / dz = - a_grav / cs^2
 * where 
 *    a_grav = (GM * z) / (R^2 + z^2)^(3/2).
 *  
 */
void setup_hydrostatic_matrix(const Star& star, const Grid& g, const Field<double>& cs2, 
                               Field<double>& rho) {
    dim3 threads(32, 32, 1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32, (g.NR+2*g.Nghost+31)/32 ) ;

    setup_hydrostatic_maxtrix_device<<<blocks, threads>>>(star.GM, g, cs2, rho) ;
    check_CUDA_errors("setup_hydrostatic_maxtrix") ;
}

__global__ void convert_pressure_to_density_device(GridRef g, FieldConstRef<double> cs2, 
                                                   FieldRef<double> rho) {
                                            
    int j = threadIdx.x + blockIdx.x*blockDim.x ;
    int i = threadIdx.y + blockIdx.y*blockDim.y ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) 
        rho[rho.index(i, j)] /= cs2[cs2.index(i, j)] ;
}


void convert_pressure_to_density(const Grid& g, const Field<double>& cs2,
                                 Field<double>& rho) {

    dim3 threads(32, 32, 1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32, (g.NR+2*g.Nghost+31)/32 ) ;

    convert_pressure_to_density_device<<<blocks, threads>>>(g, cs2, rho) ;
    check_CUDA_errors("convert_pressure_to_density_device") ;
}


__global__ void normalize_density_device(GridRef g, FieldRef<double> rho, 
                                         const double* Sigma, const double *norm) {

    int j = threadIdx.x + blockIdx.x*blockDim.x ;
    int i = threadIdx.y + blockIdx.y*blockDim.y ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        double Area = 0.5*(g.Re(i+1)*g.Re(i+1) - g.Re(i)*g.Re(i)) ;  
        rho(i, j) *= 0.5 * Sigma[i] * Area / norm[i] ; 
        rho(i, j) = max(rho(i, j), 1e-200);
    }
    __syncthreads() ;

    // Finally set the density in the ghost cell.
    if (i < g.NR + 2*g.Nghost && j < g.Nghost)
        rho(i, j) = rho(i, 2*g.Nghost-j-1) ;
}

void normalize_density(const Grid& g, Field<double>& rho, 
                       const CudaArray<double>& Sigma, const CudaArray<double>& norm) {
  
    dim3 threads(32, 32, 1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32, (g.NR+2*g.Nghost+31)/32 ) ;

    normalize_density_device<<<blocks, threads>>>(g, rho, Sigma.get(), norm.get()) ;
    check_CUDA_errors("normalize_density") ;
}


void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<double>& rho, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma) {

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;    
}

__global__ void _rho_from_wg(GridRef g, FieldRef<double> rho, FieldRef<Prims> w_g) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            rho(i,j) = w_g(i,j).rho;
        }
    }

}
__global__ void _wg_from_rho(GridRef g, FieldRef<double> rho, FieldRef<Prims> w_g, double gasfloor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            w_g(i,j).rho = rho(i,j) + gasfloor;
        }
    }

}

__global__ void _wg_from_rho(GridRef g, FieldRef<double> rho, FieldRef<Prims> w_g, double gasfloor, MoleculeRef mol) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            mol.vap(i,j) *= (rho(i,j) + gasfloor) / w_g(i,j).rho;
            w_g(i,j).rho = rho(i,j) + gasfloor;
        }
    }

}

// __global__ void _check_dust(GridRef g, FieldRef<double> rho, FieldRef<Quants> w_g, Field3DRef<Quants> q_d) {

//     int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
//     int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
//     int istride = gridDim.x * blockDim.x ;
//     int jstride = gridDim.y * blockDim.y ;

//     for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
//         for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
//             if (w_g(i,j).rho/rho(i,j) > 100. || w_g(i,j).rho/rho(i,j) < 0.01) {
//                 for (int k=0; k<q_d.Nd; k++) {
//                     q_d(i,j,k).rho = 1.e-40;
//                     q_d(i,j,k).mom_R = 0.;
//                     q_d(i,j,k).amom_phi = 0.;
//                     q_d(i,j,k).mom_Z = 0.;
//                 }
//             }
//         }
//     }

// }

void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;

    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor);  
}

void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, Field3D<Prims>& q_d, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;

    // _check_dust<<<blocks,threads>>>(g, rho, w_g, q_d);

    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor);  
}

__global__
void _calc_Z_new(GridRef g, FieldRef<double> Sig0, FieldRef<double> Sig1, FieldRef<double> Z_new) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {

            int j_r = g.Nphi+2*g.Nghost-1-j;

            int j_interp = g.Nphi+g.Nghost-2;
            for (int jint=g.Nghost; jint<g.Nphi+g.Nghost; jint++) {
                if (Sig0(i,j) < Sig1(i,jint)) {
                    j_interp = jint-1;
                    break;
                }
            }
            // if (i==28) {printf("%d %d\n", j, j_interp);}

            Z_new(i,j) = g.Zc(i,g.Nphi+2*g.Nghost-1-j_interp) + (Sig0(i,j)-Sig1(i,j_interp)) * (g.Zc(i,g.Nphi+2*g.Nghost-1-(j_interp+1)) - g.Zc(i,g.Nphi+2*g.Nghost-1-j_interp))/(Sig1(i,j_interp+1)-Sig1(i,j_interp));

            
            // if ((Z_new(i,j)) > 10.*g.Zc(i,g.Nphi+2*g.Nghost-1) || Z_new(i,j)<0.) {Z_new(i,j) = g.Zc(i,j_r);}

        }
    }
}

__global__
void _calc_Sigvap_new(GridRef g, FieldRef<double> Sig0, FieldRef<double> Sig1, FieldRef<double> Z_new) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {
            int j_r = g.Nphi+2*g.Nghost-1-j;

            int j_interp = g.Nphi+g.Nghost-2;
            for (int jint=g.Nghost; jint<g.Nphi+g.Nghost; jint++) {
                if (g.Zc(i,j_r) > Z_new(i,jint)) {
                    j_interp = jint-1;
                    break;
                }
            }
            // if (i==28) {printf("%d %d\n", j, j_interp);}

            Sig1(i,j) = exp(log(Sig0(i,j_interp)) + (g.Zc(i,j_r)-Z_new(i,j_interp)) * (log(Sig0(i,j_interp+1))-log(Sig0(i,j_interp))) / (Z_new(i,j_interp+1)-Z_new(i,j_interp)));
            

            // int j_r = g.Nphi+2*g.Nghost-1-j;
            // if (j<g.Nphi+g.Nghost-1) {
            //     Sig1(i,j) = Sig0(i,j_r) + (g.Zc(i,j)-Z_new(i,j)) * (Sig0(i,j_r-1)-Sig0(i,j_r)) / (Z_new(i,j+1)-Z_new(i,j));
            // }
            // else {
            //     Sig1(i,j) = Sig0(i,j_r) + (g.Zc(i,j)-Z_new(i,j)) * (Sig0(i,j_r)-Sig0(i,j_r+1)) / (Z_new(i,j)-Z_new(i,j-1));
            // }
        
        }
    }
}

__global__
void _calc_Sigma(GridRef g, FieldRef<double> rho, FieldRef<double> Sig) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {

            Sig(i,g.Nphi+2*g.Nghost-1-j) = (rho(i,j)*g.dZe(i,j));

        }
    }
}
__global__
void _calc_CDF(GridRef g, FieldRef<double> CDF) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int istride = gridDim.x * blockDim.x ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=g.Nghost; j<g.Nphi+g.Nghost; j++) {

            CDF(i,j+1) = exp(log(CDF(i,j+1)) + log(1.+CDF(i,j)/CDF(i,j+1)));

        }
    }
}

__global__
void _calc_rho_vap(GridRef g, FieldRef<Prims> w_g, FieldRef<double> Sig, FieldRef<double> vap, double gas_floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int istride = gridDim.x * blockDim.x ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=g.Nghost; j<g.Nphi+g.Nghost; j++) {

            if (w_g(i,j).rho < 10.*gas_floor) {
                vap(i,j) = gas_floor;
            }
            else {
                vap(i,j) = max((Sig(i,g.Nphi+2*g.Nghost-1-j)-Sig(i,g.Nphi+2*g.Nghost-1-(j+1)))/g.dZe(i,j),gas_floor);
            }
            
            // vap(i,j) = max((Sig(i,j)-Sig(i,j+1))/g.dZe(i,j),gas_floor);
            // vap(i,j) = max(exp(log(Sig(i,j))+log(1.-Sig(i,j-1)/Sig(i,j)))/g.dZe(i,j),gas_floor);

        }
    }
}

void compute_hydrostatic_equilibrium2(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, Molecule& mol, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    Field<double> SigmaZ0 = create_field<double>(g);
    set_all(g, SigmaZ0, 0.);
    _calc_Sigma<<<blocks,threads>>>(g, rho, SigmaZ0);
    // _calc_CDF<<<1,1024>>>(g, SigmaZ0);
    Reduction::scan_Z_sum(g, SigmaZ0);
    // hipDeviceSynchronize();
    // for (int i=0; i<g.Nphi+2*g.Nghost; i++) {
    //     std::cout << SigmaZ0(28,i) << ",";
    // }
    // std::cout << "\n";
    

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;

    Field<double> SigmaZ = create_field<double>(g);
    set_all(g, SigmaZ, 0.);
    _calc_Sigma<<<blocks,threads>>>(g, rho, SigmaZ);
    // _calc_CDF<<<1,1024>>>(g, SigmaZ);
    Reduction::scan_Z_sum(g, SigmaZ);
    // hipDeviceSynchronize();
    // for (int i=0; i<g.Nphi+2*g.Nghost; i++) {
    //     std::cout << SigmaZ(28,i) << ",";
    // }
    // std::cout << "\n";
    

    Field<double> Z_new = create_field<double>(g);
    _calc_Z_new<<<blocks,threads>>>(g, SigmaZ0, SigmaZ, Z_new); 
    // hipDeviceSynchronize();
    // for (int i=0; i<g.Nphi+2*g.Nghost; i++) {
    //     std::cout << Z_new(28,i) << ",";
    // }
    // std::cout << "\n";
    // for (int i=0; i<g.Nphi+2*g.Nghost; i++) {
    //     std::cout << g.Zc(28,g.Nphi+2*g.Nghost-1-i) << ",";
    // }
    // std::cout << "\n";

    set_all(g, SigmaZ0, 0.);
    _calc_Sigma<<<blocks,threads>>>(g, mol.vap, SigmaZ0);
    Reduction::scan_Z_sum(g, SigmaZ0);
    // _calc_CDF<<<1,1024>>>(g, SigmaZ0);

    _calc_Sigvap_new<<<blocks,threads>>>(g, SigmaZ0, SigmaZ, Z_new);

    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor);  
    _calc_rho_vap<<<1,1024>>>(g, w_g, SigmaZ, mol.vap, gasfloor);
}

void compute_hydrostatic_equilibrium(const Star& star, const Grid& g, Field<Prims>& w_g, 
                                     const Field<double>& cs2, const CudaArray<double>& Sigma, Molecule& mol, double gasfloor) {
    
    Field<double> rho = create_field<double>(g);
    
    dim3 threads(32,32,1);
    dim3 blocks((g.NR+2*g.Nghost+31)/32, (g.Nphi+2*g.Nghost+31)/32 );

    _rho_from_wg<<<blocks,threads>>>(g, rho, w_g);

    if (g.Nghost > 64) {
        std::string msg = 
            "compute_hydrostatic_equilibrium only works for Nghost <= 16" ;
        throw std::invalid_argument(msg);
    }

    CodeTiming::BlockTimer timing_block = 
        timer->StartNewTimer("compute_hydrostatic_equilibrium") ;

    // Step 1: Setup the finite difference factors for hydrostatic equilibrium
    setup_hydrostatic_matrix(star, g, cs2, rho) ;
    
    // Step 2: Solve the relation using parallel scan
    Reduction::scan_Z_mul(g, rho) ;
    convert_pressure_to_density(g, cs2, rho) ;

    // Step 3 compute the normalizations:
    zero_midplane_boundary(g, rho) ;

    CudaArray<double> norm = make_CudaArray<double>(g.NR + 2*g.Nghost) ;
    Reduction::volume_integrate_Z(g, rho, norm) ;

    // Step 4: Multiply rho by normalization 
    normalize_density(g, rho, Sigma, norm) ;


    _wg_from_rho<<<blocks,threads>>>(g, rho, w_g, gasfloor, mol);  
}