#include "hip/hip_runtime.h"
#include <stdexcept>

#include "timing.h"
#include "utils.h"
#include "coagulation/coagulation.h"
#include "coagulation/fragments.h"
#include "coagulation/integration.h"
#include "dustdynamics.h"

#include <iostream>

template<typename T>
__global__ void _compute_ytot(GridRef g, Field3DConstRef<double> y, 
                              FieldRef<double> yscale, double scale, FieldRef<T> wg) {

    int j = threadIdx.x + blockIdx.x * blockDim.x ;
    int i = threadIdx.y + blockIdx.y * blockDim.y ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        double res = 0 ;
        for (int k=0; k<y.Nd; k++)
            res += y(i,j,k) ;
        if (res > y.Nd*10*1e-40*wg(i,j)[0]) {
            yscale(i,j) = (res+1e-100)*scale ;
        }
        else {
            yscale(i,j) = 1.;
        }
    }
}

/* specialization for double */
__global__ void _compute_ytot(GridRef g, Field3DConstRef<double> y, 
                              FieldRef<double> yscale, double scale, FieldRef<double> wg) {

    int j = threadIdx.x + blockIdx.x * blockDim.x ;
    int i = threadIdx.y + blockIdx.y * blockDim.y ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        double res = 0 ;
        for (int k=0; k<y.Nd; k++)
            res += y(i,j,k) ;
        if (res > y.Nd*10*1e-40*wg(i,j)) {
            yscale(i,j) = (res+1e-100)*scale ;
        }
        else {
            yscale(i,j) = 1.;
        }
    }
}

// Compute the maximum error scaled in each block. 
// The result is stored in the errtot(i,j) corresponding to threadIdx.{x,y} = 0.
__global__ void _compute_error_norm(GridRef g, 
                                    Field3DConstRef<double> y, Field3DConstRef<double> ynew, 
                                    FieldConstRef<double> yabs, double rel_tol,
                                    Field3DConstRef<double> err, FieldRef<double> errtot) {

    int j = threadIdx.x + blockIdx.x * blockDim.x ;
    int i = threadIdx.y + blockIdx.y * blockDim.y ;

    // Get the total scaled error for each cell.
    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        double res = 0 ;
        double scale ;
        for (int k=0; k<y.Nd; k++) {
            scale = yabs(i,j) + max(abs(y(i,j,k)), abs(ynew(i,j,k))) * rel_tol ;
            res += err(i,j,k)*err(i,j,k) / (scale*scale) ;
        }
        errtot(i,j) = res ;
    }
    __syncthreads() ;

    // Compute the max error over each cell
    //   1. Reduce over y
    int size = blockDim.x / 2 ;
    while (size > 0) {
        if (threadIdx.x < size && (i < g.NR + 2*g.Nghost && j + size < g.Nphi + 2*g.Nghost))
            errtot(i,j) = max(errtot(i,j), errtot(i, j+size)) ;
        
        size /= 2 ;
        __syncthreads() ;
    }

    //   2. Reduce over x
    size = blockDim.y / 2 ;
    if (blockIdx.x * blockDim.x < g.Nphi + 2*g.Nghost) {        
        while (size > 0) {
            if (threadIdx.x == 0 && threadIdx.y < size && i + size < g.NR + 2*g.Nghost)
                errtot(i,j) = max(errtot(i,j), errtot(i+size, j)) ;

            size /= 2 ;
            __syncthreads() ;
        }
    }
} 
__global__ void _compute_error_norm_debug(GridRef g, 
                                    Field3DConstRef<double> y, Field3DConstRef<double> ynew, 
                                    FieldConstRef<double> yabs, double rel_tol,
                                    Field3DConstRef<double> err, FieldRef<double> errtot, Field3DRef<int> idxs) {

    int j = threadIdx.x + blockIdx.x * blockDim.x ;
    int i = threadIdx.y + blockIdx.y * blockDim.y ;

    // Get the total scaled error for each cell.
    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        double res = 0 ;
        double scale ;
        for (int k=0; k<y.Nd; k++) {
            scale = yabs(i,j) + max(abs(y(i,j,k)), abs(ynew(i,j,k))) * rel_tol ;
            res += err(i,j,k)*err(i,j,k) / (scale*scale) ;
        }
        errtot(i,j) = res ;
        idxs(i,j,0) = i ;
        idxs(i,j,1) = j ;
    }
    __syncthreads() ;

    // Compute the max error over each cell
    //   1. Reduce over y
    int size = blockDim.x / 2 ;
    while (size > 0) {
        if (threadIdx.x < size && (i < g.NR + 2*g.Nghost && j + size < g.Nphi + 2*g.Nghost)) {
            if ( errtot(i, j+size) > errtot(i,j)) {
                errtot(i,j) = errtot(i, j+size);
                idxs(i,j,0) = idxs(i,j+size, 0);
                idxs(i,j,1) = idxs(i,j+size, 1);
            } 
        }
             
        
        size /= 2 ;
        __syncthreads() ;
    }

    //   2. Reduce over x
    size = blockDim.y / 2 ;
    if (blockIdx.x * blockDim.x < g.Nphi + 2*g.Nghost) {        
        while (size > 0) {
            if (threadIdx.x == 0 && threadIdx.y < size && i + size < g.NR + 2*g.Nghost) {
                if ( errtot(i+size, j) > errtot(i,j)) {
                    errtot(i,j) = errtot(i+size, j);
                    idxs(i,j,0) = idxs(i+size, j, 0) ;
                    idxs(i,j,1) = idxs(i+size, j, 1) ;
                } 
            }
                 

            size /= 2 ;
            __syncthreads() ;
        }
    }
} 

template<typename T>
double TimeIntegration::take_step(Grid& g, Field3D<double>& y, Field<T>& wg, double& dtguess) const {

    CodeTiming::BlockTimer block =
        timer->StartNewTimer("TimeIntegation::take_step");
  
    Field3D<double> ynew  = create_field3D<double>(g, y.Nd) ;
    Field3D<double> error = create_field3D<double>(g, y.Nd) ;

    Field<double> yabs    = create_field<double>(g) ;
    Field<double> err_tot = create_field<double>(g) ;

    
    double dt ;
    if (dtguess > 0) {
        // Use guess provided
        dt = dtguess ;
    }
    else {
        dt = 1 ;
    }

    dim3 threads(32,32,1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32,(g.NR+2*g.Nghost+31)/32,1) ;

    // Compute the total density for the error estimation
    _compute_ytot<<<blocks,threads>>>(g, y, yabs, _abs_tol, FieldRef<T>(wg)) ; 
    check_CUDA_errors("_compute_ytot") ;
      
    bool success = false ;

    while (not success) {
        if (dt == 0)
            throw std::runtime_error("Error time-step of zero was assigned");
          
        do_step(dt, g, y, ynew, error) ;

        // Compute the normalized error
        _compute_error_norm<<<blocks,threads>>>(g, y, ynew, yabs, _rel_tol, 
                                                error, err_tot) ;
        check_CUDA_errors("_compute_error_norm") ;

        double err_norm = 0 ;
        for (int i=0; i < g.NR + 2*g.Nghost; i += 32)
            for (int j=0; j < g.Nphi + 2*g.Nghost; j += 32)
                err_norm = std::max(err_norm, err_tot(i,j)) ;

        if (err_norm < 1) {
            success = true ;

            dtguess = dt * std::min(_MAX_FACTOR,
                                std::max(1., _SAFETY * std::pow(err_norm, -0.5 / _order)));
        } else {
            dt  = dt * std::max(_MIN_FACTOR, _SAFETY * std::pow(err_norm, -0.5 / _order)) ;
        }
    }

    copy_field(g, ynew, y) ;

    return dt ;
}
template<typename T>
double TimeIntegration::take_step_debug(Grid& g, Field3D<double>& y, Field<T>& wg, double& dtguess, int* idxs) const {

    CodeTiming::BlockTimer block =
        timer->StartNewTimer("TimeIntegation::take_step");
  
    Field3D<double> ynew  = create_field3D<double>(g, y.Nd) ;
    Field3D<double> error = create_field3D<double>(g, y.Nd) ;

    Field<double> yabs    = create_field<double>(g) ;
    Field<double> err_tot = create_field<double>(g) ;

    Field3D<int> idxgrid = create_field3D<int>(g, 2);

    
    double dt ;
    if (dtguess > 0) {
        // Use guess provided
        dt = dtguess ;
    }
    else {
        dt = 1 ;
    }

    dim3 threads(32,32,1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32,(g.NR+2*g.Nghost+31)/32,1) ;

    // Compute the total density for the error estimation
    _compute_ytot<<<blocks,threads>>>(g, y, yabs, _abs_tol, FieldRef<T>(wg)) ; 
    check_CUDA_errors("_compute_ytot") ;
      
    bool success = false ;

    while (not success) {
        if (dt == 0)
            throw std::runtime_error("Error time-step of zero was assigned");
          
        do_step(dt, g, y, ynew, error) ;

        // Compute the normalized error
        _compute_error_norm_debug<<<blocks,threads>>>(g, y, ynew, yabs, _rel_tol, 
                                                error, err_tot, idxgrid) ;
        check_CUDA_errors("_compute_error_norm") ;

        double err_norm = 0 ;
        for (int i=0; i < g.NR + 2*g.Nghost; i += 32)
            for (int j=0; j < g.Nphi + 2*g.Nghost; j += 32)
                if (err_tot(i,j) > err_norm) {
                    err_norm = std::max(err_norm, err_tot(i,j)) ;
                    idxs[0] = idxgrid(i,j,0);
                    idxs[1] = idxgrid(i,j,1);
                }

        if (err_norm < 1) {
            success = true ;

            dtguess = dt * std::min(_MAX_FACTOR,
                                std::max(1., _SAFETY * std::pow(err_norm, -0.5 / _order)));
        } else {
            dt  = dt * std::max(_MIN_FACTOR, _SAFETY * std::pow(err_norm, -0.5 / _order)) ;
        }
    }

    copy_field(g, ynew, y) ;

    return dt ;
}

template<typename T>
__global__ void _copy_rho_forwards(GridRef g, Field3DRef<T> ws, FieldRef<T> wg, Field3DRef<double> rhos, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<ws.Nd; k+=kstride) { 
                rhos(i,j,k) = max(ws(i,j,k)[0]-floor*wg(i,j)[0], 0.);
            }
        }
    }
}

/* Specialization for type double */
__global__ void _copy_rho_forwards(GridRef g, Field3DRef<double> ws, FieldRef<double> wg, Field3DRef<double> rhos, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<ws.Nd; k+=kstride) { 
                rhos(i,j,k) = max(ws(i,j,k)-floor*wg(i,j), 0.);
            }
        }
    }
}

template<typename T>
__global__ void _copy_rho_backwards(GridRef g, Field3DRef<T> ws, FieldRef<T> wg, Field3DRef<double> rhos, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<ws.Nd; k+=kstride) { 
                ws(i,j,k)[0] = rhos(i,j,k) + floor*wg(i,j)[0]; 
            }
        }
    }
}

/* Specialization for type double */
__global__ void _copy_rho_backwards(GridRef g, Field3DRef<double> ws, FieldRef<double> wg, Field3DRef<double> rhos, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<ws.Nd; k+=kstride) { 
                ws(i,j,k) = rhos(i,j,k) + floor*wg(i,j); 
            }
        }
    }
}

double calc_mass(Grid& g, Field3D<double>& q) {

    double mass=0;

    for (int i=g.Nghost; i<g.NR+g.Nghost; i++) {
        for (int j=g.Nghost; j<g.Nphi+g.Nghost; j++) {
            for (int k=0; k<q.Nd; k++) {
                mass += 4*M_PI* q(i,j,k) * g.volume(i,j);
            }
        }
    }

    return mass;
}
double calc_mass_cell(Grid& g, Field3D<double>& q) {

    double mass=0;

    for (int k=0; k<q.Nd; k++) {
        mass += q(52,2,k);
    }


    return mass;
}

template<typename T>
int TimeIntegration::integrate(Grid& g, Field3D<T>& ws, Field<T>& wg, double tmax, double& dt_coag, double floor) const {
    double dt = dt_coag ;
    if (dt_coag < tmax && dt_coag > _SAFETY*tmax)
        dt /= 2 ;

    double t = 0 ;

    Field3D<double> rhos = create_field3D<double>(g, ws.Nd);
    set_all(g, rhos, 0.);

    dim3 threads(16,8,8);
    dim3 blocks((g.NR + 2*g.Nghost+15)/16,(g.Nphi + 2*g.Nghost+7)/8, (ws.Nd+7)/8) ;

    _copy_rho_forwards<<<blocks,threads>>>(g, Field3DRef<T>(ws), FieldRef<T>(wg), rhos, floor);
    hipDeviceSynchronize();
    int count = 0;

    while (t < tmax) {
        dt = std::min(dt, tmax-t) ;
        t += take_step(g, rhos, wg, dt) ;
        count += 1;
        if (_verbose && (count%100) == 0) {
            std::cout << "Coagulation Steps = " << count << ", dt_coag = " << dt/year << " years, t = " << t/year << " years \n";
        }
    }
    if (_verbose) 
        std::cout << "Coagulation Steps = " << count << ", dt_coag = " << dt/year << " years, t = " << t/year << " years \n";
    
    dt_coag = dt;

    _copy_rho_backwards<<<blocks,threads>>>(g, Field3DRef<T>(ws), FieldRef<T>(wg), rhos, floor);

    return count ;
}

template<typename T>
int TimeIntegration::integrate_debug(Grid& g, Field3D<T>& ws, Field<T>& wg, double tmax, double& dt_coag, double floor) const {
    double dt = dt_coag ; 
    if (dt_coag < tmax && dt_coag > _SAFETY*tmax)
        dt /= 2 ;
    double t = 0 ;

    Field3D<double> rhos = create_field3D<double>(g, ws.Nd);
    set_all(g, rhos, 0.);
    
    dim3 threads(16,8,8);
    dim3 blocks((g.NR + 2*g.Nghost+15)/16,(g.Nphi + 2*g.Nghost+7)/8, (ws.Nd+7)/8) ;

    _copy_rho_forwards<<<blocks,threads>>>(g, Field3DRef<T>(ws), FieldRef<T>(wg), rhos, floor);
    hipDeviceSynchronize();
    int count = 0;
    int idxs[2] = {0,0};

    while (t < tmax) {


        dt = std::min(dt, tmax-t) ;
        t += take_step_debug(g, rhos, wg, dt, idxs) ;
        if (!(count%100) && count) {
            std::cout << "Count = " << count << ", dt_coag = " << dt/year << " years, t = " << t/year << " years \n";
            std::cout << "i index = " << idxs[0] << ", j index = " << idxs[1] << "\n";
        }
        count += 1;
        // printf("%1.12g %1.12g %g\n", calc_mass(g,rhos), calc_mass_cell(g,rhos), dt);
        // printf("%1.12g\n", calc_mass_cell(g,rhos));
    }
    std::cout << "Count = " << count << ", dt_coag = " << dt/year << " years, t = " << t/year << " years \n";
    std::cout << "i index = " << idxs[0] << ", j index = " << idxs[1] << "\n";

    dt_coag = dt;

    _copy_rho_backwards<<<blocks,threads>>>(g, Field3DRef<T>(ws), FieldRef<T>(wg), rhos, floor);

    return count ;
}

__global__ void _Rk2_update1(GridRef g, Field3DConstRef<double> y, 
                             Field3DConstRef<double> rate, double dt, Field3DRef<double> y_new) {
   
    int i = threadIdx.z + blockIdx.z * blockDim.z ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.x + blockIdx.x * blockDim.x ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {

        for (/**/; k < y.Nd; k += blockDim.x*gridDim.x) 
            y_new(i,j,k) = max(y(i,j,k) + rate(i,j,k) * dt, 0.0) ;
    }
}

// Compute Heun's method update. 
//   Note that error is used as the rate on input
__global__ void _Rk2_update2(GridRef g, Field3DConstRef<double> y, 
                             double dt, Field3DRef<double> y_new, Field3DRef<double> error) {

    int i = threadIdx.z + blockIdx.z * blockDim.z ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.x + blockIdx.x * blockDim.x ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        for (/**/; k < y.Nd; k += blockDim.x*gridDim.x) {
            double ys = y_new(i,j,k) ;
            double y1 = max(0.5*(ys + y(i,j,k) + error(i,j,k) * dt), 0.);
        
            y_new(i,j,k) = y1 ;
            error(i,j,k) = y1-ys ;
        }
    }
}


template<class Rate>
void Rk2Integration<Rate>::do_step(double dt, Grid& g, const Field3D<double>& y,
                                   Field3D<double>& ynew, Field3D<double>& error) const {

    CodeTiming::BlockTimer block =
        timer->StartNewTimer("Rk2Integration::do_step") ;

    Field3D<double>& rate = error ;

    // Compute the rate
    this->operator()(y, rate) ;

    dim3 threads(32,8,4) ;
    dim3 blocks((y.Nd+31)/32, (g.Nphi +2*g.Nghost + 7)/8, (g.NR + 2*g.Nghost + 3)/4);

    // 1st guess (Euler's method)
    _Rk2_update1<<<blocks, threads>>>(g, y, rate, dt, ynew) ;
    check_CUDA_errors("_Rk2_update1") ;

    // Compute the rate, correction, and error (Heun's method)
    this->operator()(ynew, rate) ;

    _Rk2_update2<<<blocks, threads>>>(g, y, dt, ynew, error) ;
    check_CUDA_errors("_Rk2_update2") ;

}

__global__ void _BS32_update1(GridRef g, Field3DConstRef<double> y, 
                             Field3DConstRef<double> rate, double dt, Field3DRef<double> y_new) {
   
    int i = threadIdx.z + blockIdx.z * blockDim.z ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.x + blockIdx.x * blockDim.x ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {

        for (/**/; k < y.Nd; k += blockDim.x*gridDim.x) 
            y_new(i,j,k) = max(y(i,j,k) + 0.5*rate(i,j,k) * dt, 0.0) ;
    }
}
__global__ void _BS32_update2(GridRef g, Field3DConstRef<double> y, Field3DConstRef<double> rate2, double dt, Field3DRef<double> y_new) {
   
    int i = threadIdx.z + blockIdx.z * blockDim.z ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.x + blockIdx.x * blockDim.x ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {

        for (/**/; k < y.Nd; k += blockDim.x*gridDim.x) 
            y_new(i,j,k) = max(y(i,j,k) + dt*(3./4. * rate2(i,j,k)), 0.0) ;
    }
}

__global__ void _BS32_update3(GridRef g, Field3DConstRef<double> y, 
                             Field3DConstRef<double> rate1, Field3DConstRef<double> rate2, Field3DConstRef<double> rate3, double dt, Field3DRef<double> y_new) {
   
    int i = threadIdx.z + blockIdx.z * blockDim.z ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.x + blockIdx.x * blockDim.x ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {

        for (/**/; k < y.Nd; k += blockDim.x*gridDim.x) 
            y_new(i,j,k) = max(y(i,j,k) + dt*(2./9. * rate1(i,j,k) + 1./3. * rate2(i,j,k) + 4./9. * rate3(i,j,k)), 0.0) ;
    }
}


__global__ void _BS32_update4(GridRef g, Field3DConstRef<double> y, 
                             double dt, Field3DRef<double> y_new, Field3DConstRef<double> rate1, Field3DConstRef<double> rate2, 
                             Field3DConstRef<double> rate3, Field3DConstRef<double> rate4, Field3DRef<double> error) {

    int i = threadIdx.z + blockIdx.z * blockDim.z ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.x + blockIdx.x * blockDim.x ;

    if (i < g.NR + 2*g.Nghost && j < g.Nphi + 2*g.Nghost) {
        for (/**/; k < y.Nd; k += blockDim.x*gridDim.x) {

            double ys = y_new(i,j,k);
            double y1 = max(y(i,j,k) + dt*(7./24. * rate1(i,j,k) + 1./4. * rate2(i,j,k) + 1./3. * rate3(i,j,k) + 1./8. * rate4(i,j,k)), 0.);
        
            y_new(i,j,k) = y1 ;
            error(i,j,k) = y1-ys ;
        }
    }
}


template<class Rate>
void BS32Integration<Rate>::do_step(double dt, Grid& g, const Field3D<double>& y,
                                   Field3D<double>& ynew, Field3D<double>& error) const {
                                
    // Bogacki-Shampine embedded Runge-Kutta 3(2) method: https://www.sciencedirect.com/science/article/pii/0893965989900797
    
    CodeTiming::BlockTimer block =
        timer->StartNewTimer("BS32Integration::do_step") ;

    Field3D<double> k1 = create_field3D<double>(g, y.Nd);
    Field3D<double> k2 = create_field3D<double>(g, y.Nd);
    Field3D<double> k3 = create_field3D<double>(g, y.Nd);
    Field3D<double> k4 = create_field3D<double>(g, y.Nd);

    dim3 threads(32,8,4) ;
    dim3 blocks((y.Nd+31)/32, (g.Nphi +2*g.Nghost + 7)/8, (g.NR + 2*g.Nghost + 3)/4);

    this->operator()(y, k1) ;

    _BS32_update1<<<blocks, threads>>>(g, y, k1, dt, ynew) ;

    this->operator()(ynew, k2) ;

    _BS32_update2<<<blocks, threads>>>(g, y, k2, dt, ynew) ;

    this->operator()(ynew, k3) ;

    _BS32_update3<<<blocks, threads>>>(g, y, k1, k2, k3, dt, ynew) ;

    this->operator()(ynew, k4) ;

    _BS32_update4<<<blocks, threads>>>(g, y, dt, ynew, k1, k2, k3, k4, error) ;

}

// Ice tracer coagulation


__global__
void _combine_rho_tr(GridRef g, Field3DRef<double> rhos, Field3DRef<double> tr, Field3DRef<double> rho_tr) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            for (int k=kidx; k<rhos.Nd; k+=kstride) { 
                rho_tr(i,j,k) = rhos(i,j,k);
                rho_tr(i,j,k+rhos.Nd) = tr(i,j,k);
            }
        }
    }
}

__global__
void _decombine_rho_tr(GridRef g, Field3DRef<double> rhos, Field3DRef<double> tr, Field3DRef<double> rho_tr) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx; i<g.NR+2*g.Nghost; i+=istride) {
        for (int j=jidx; j<g.Nphi+2*g.Nghost; j+=jstride) { 
            for (int k=kidx; k<rhos.Nd; k+=kstride) { 
                rhos(i,j,k) = rho_tr(i,j,k);
                tr(i,j,k) = rho_tr(i,j,k+rhos.Nd);
            }
        }
    }
}

template<typename T>
double TimeIntegration::take_step_tracers(Grid& g, Field3D<double>& y, Field<T>& wg, double& dtguess, Field3D<double>& tracers, int* idxs) const {

    CodeTiming::BlockTimer block =
        timer->StartNewTimer("TimeIntegation::take_step");
  
    Field3D<double> ynew  = create_field3D<double>(g, 2*y.Nd) ;
    Field3D<double> error = create_field3D<double>(g, 2*y.Nd) ;
    Field3D<double> ywtr = create_field3D<double>(g, 2*y.Nd) ;

    Field<double> yabs    = create_field<double>(g) ;
    Field<double> err_tot = create_field<double>(g) ;

    Field3D<int> idxgrid = create_field3D<int>(g, 2);

    double dt ;
    if (dtguess > 0) {
        // Use guess provided
        dt = dtguess ;
    }
    else {
        dt = 1 ;
    }

    dim3 threads(32,32,1) ;
    dim3 blocks((g.Nphi+2*g.Nghost+31)/32,(g.NR+2*g.Nghost+31)/32,1) ;

    // Compute the total density for the error estimation
    _compute_ytot<<<blocks,threads>>>(g, y, yabs, _abs_tol, FieldRef<T>(wg)) ; 
    check_CUDA_errors("_compute_ytot") ;
      
    bool success = false ;

    _combine_rho_tr<<<blocks,threads>>>(g, y, tracers, ywtr);

    while (not success) {
        if (dt == 0)
            throw std::runtime_error("Error time-step of zero was assigned");
          
        do_step(dt, g, ywtr, ynew, error) ;

        // Compute the normalized error
        _compute_error_norm_debug<<<blocks,threads>>>(g, y, ynew, yabs, _rel_tol, 
                                                error, err_tot, idxgrid) ;
        check_CUDA_errors("_compute_error_norm") ;

        double err_norm = 0 ;
        for (int i=0; i < g.NR + 2*g.Nghost; i += 32) {
            for (int j=0; j < g.Nphi + 2*g.Nghost; j += 32) {
                if (err_tot(i,j) > err_norm) {
                    err_norm = std::max(err_norm, err_tot(i,j)) ;
                    idxs[0] = idxgrid(i,j,0);
                    idxs[1] = idxgrid(i,j,1);
                }
            }
        }

        if (err_norm < 1) {
            success = true ;

            dtguess = dt * std::min(_MAX_FACTOR,
                                std::max(1., _SAFETY * std::pow(err_norm, -0.5 / _order)));
        } else {
            dt  = dt * std::max(_MIN_FACTOR, _SAFETY * std::pow(err_norm, -0.5 / _order)) ;
        }
    }

    copy_field(g, ynew, ywtr) ;
    _decombine_rho_tr<<<blocks,threads>>>(g, y, tracers, ywtr);

    return dt ;
}

template<typename T>
__global__ void _remove_tr_floor(GridRef g, FieldRef<T> wg, Field3DRef<double> tr, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<tr.Nd; k+=kstride) { 
                tr(i,j,k) = max(tr(i,j,k)-1.1e-100*floor*wg(i,j)[0], 0.);
            }
        }
    }
}

template<typename T>
__global__ void _add_tr_floor(GridRef g, FieldRef<T> wg, Field3DRef<double> tr, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<tr.Nd; k+=kstride) { 
                tr(i,j,k) += 1e-100*floor*wg(i,j)[0];
            }
        }
    }
}

__global__ void _remove_tr_floor(GridRef g, FieldRef<double> wg, Field3DRef<double> tr, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<tr.Nd; k+=kstride) { 
                tr(i,j,k) = max(tr(i,j,k)-1.1e-100*floor*wg(i,j), 0.);
            }
        }
    }
}

__global__ void _add_tr_floor(GridRef g, FieldRef<double> wg, Field3DRef<double> tr, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) { 
            for (int k=kidx; k<tr.Nd; k+=kstride) { 
                tr(i,j,k) += 1e-100*floor*wg(i,j);
            }
        }
    }
}

template<typename T>
void TimeIntegration::integrate_tracers(Grid& g, Field3D<T>& ws, Field<T>& wg, Molecule& mol, double tmax, double& dt_coag, double floor) const {
    double dt = dt_coag ;
    if (dt_coag < tmax && dt_coag > _SAFETY*tmax)
        dt /= 2 ;

    double t = 0 ;

    Field3D<double> rhos = create_field3D<double>(g, ws.Nd);
    set_all(g, rhos, 0.);

    dim3 threads(16,8,8);
    dim3 blocks((g.NR + 2*g.Nghost+15)/16,(g.Nphi + 2*g.Nghost+7)/8, (ws.Nd+7)/8) ;

    _copy_rho_forwards<<<blocks,threads>>>(g, Field3DRef<T>(ws), FieldRef<T>(wg), rhos, floor);
    _remove_tr_floor<<<blocks,threads>>>(g, FieldRef<T>(wg), mol.ice, floor);
    hipDeviceSynchronize();
    int count = 0;
    int idxs[2] = {0,0};

    while (t < tmax) {
        // printf("%1.12g %1.12g %g\n", calc_mass(g,tracers), calc_mass_cell(g,tracers), dt);
        dt = std::min(dt, tmax-t) ;
        t += take_step_tracers(g, rhos, wg, dt, mol.ice, idxs) ;
        if (!(count%10)) {
            std::cout << "Count = " << count << ", dt_coag = " << dt/year << " years, t = " << t/year << " years \n";
            std::cout << "i index = " << idxs[0] << ", j index = " << idxs[1] << "\n";
        }
        count += 1;
        // if (dt < tmax/1e5) {
        //     dt_coag = dt;
        //     return;
        // }
        // printf("%1.12g %1.12g %g\n", calc_mass(g,tracers), calc_mass_cell(g,tracers), dt);
    }
    // std::cout << "Count = " << count << ", dt_coag = " << dt/year << " years, t = " << t/year << " years \n";
    // std::cout << "i index = " << idxs[0] << ", j index = " << idxs[1] << "\n";

    dt_coag = dt;

    _copy_rho_backwards<<<blocks,threads>>>(g, Field3DRef<T>(ws), FieldRef<T>(wg), rhos, floor);
    _add_tr_floor<<<blocks,threads>>>(g, FieldRef<T>(wg), mol.ice, floor);
}



template class Rk2Integration<CoagulationRate<BirnstielKernel<true>,SimpleErosion>> ;
template class Rk2Integration<CoagulationRate<BirnstielKernel<false>,SimpleErosion>> ;
template class Rk2Integration<CoagulationRate<BirnstielKernelVertInt<false>,SimpleErosion>> ;
template class Rk2Integration<CoagulationRate<BirnstielKernelVertInt<true>,SimpleErosion>> ;
template class Rk2Integration<CoagulationRate<BirnstielKernelIce<false>,SimpleErosion>> ;
template class Rk2Integration<CoagulationRate<BirnstielKernelIce<true>,SimpleErosion>> ;
template class Rk2Integration<CoagulationRate<ConstantKernel,SimpleErosion>> ;

template class BS32Integration<CoagulationRate<BirnstielKernel<true>,SimpleErosion>> ;
template class BS32Integration<CoagulationRate<BirnstielKernel<false>,SimpleErosion>> ;
template class BS32Integration<CoagulationRate<BirnstielKernelVertInt<false>,SimpleErosion>> ;
template class BS32Integration<CoagulationRate<BirnstielKernelVertInt<true>,SimpleErosion>> ;
template class BS32Integration<CoagulationRate<BirnstielKernelIce<false>,SimpleErosion>> ;
template class BS32Integration<CoagulationRate<BirnstielKernelIce<true>,SimpleErosion>> ;
template class BS32Integration<CoagulationRate<ConstantKernel,SimpleErosion>> ;


template int TimeIntegration::integrate_debug<Prims>(Grid& g, Field3D<Prims>& ws, Field<Prims>& wg, double tmax, double& dt_coag, double floor) const;
template int TimeIntegration::integrate_debug<Prims1D>(Grid& g, Field3D<Prims1D>& ws, Field<Prims1D>& wg, double tmax, double& dt_coag, double floor) const;
template int TimeIntegration::integrate_debug<double>(Grid& g, Field3D<double>& ws, Field<double>& wg, double tmax, double& dt_coag, double floor) const;

template void TimeIntegration::integrate_tracers<Prims>(Grid& g, Field3D<Prims>& ws, Field<Prims>& wg, Molecule& mol, double tmax, double& dt_coag, double floor) const;
template void TimeIntegration::integrate_tracers<Prims1D>(Grid& g, Field3D<Prims1D>& ws, Field<Prims1D>& wg, Molecule& mol, double tmax, double& dt_coag, double floor) const;
template void TimeIntegration::integrate_tracers<double>(Grid& g, Field3D<double>& ws, Field<double>& wg, Molecule& mol, double tmax, double& dt_coag, double floor) const;

template int TimeIntegration::integrate<Prims>(Grid& g, Field3D<Prims>& ws, Field<Prims>& wg, double tmax, double& dt_coag, double floor) const;
template int TimeIntegration::integrate<Prims1D>(Grid& g, Field3D<Prims1D>& ws, Field<Prims1D>& wg, double tmax, double& dt_coag, double floor) const;
template int TimeIntegration::integrate<double>(Grid& g, Field3D<double>& ws, Field<double>& wg, double tmax, double& dt_coag, double floor) const;
