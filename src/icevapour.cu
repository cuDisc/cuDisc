#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cassert>

#include "icevapour.h"
#include "dustdynamics.h"
#include "constants.h"

struct ChemRate {
    double rate;
    double jac;
} ;

/*  Implicit scheme

Solve system:

    drho_vap/dt = -sum(R_a,n) rho_vap + sum(R_d,n rho_ice,n)

    drho_ice,n/dt = R_a,n rho_vap - R_d,n rho_ice,n

*/

__host__ __device__
double nu_i(MoleculeRef mol, double N_s) {

    double nu_0 = std::sqrt(2*N_s*k_B / (m_H * M_PI*M_PI));

    return nu_0 * mol.T_bind/(mol.m_mol/m_H);
}

__host__ __device__
double n_ice_max(double N_s, Field3DRef<Prims>& W, const RealType* a, const RealType* m, int i, int j, int k) {

    return 4.*M_PI * a[k] * a[k] * N_s * W(i,j,k).rho / m[k];
}

__host__ __device__
ChemRate R_d_jac(MoleculeRef mol, Field3DRef<double> ice_grain, double N_s, FieldConstRef<double>& T, 
                    Field3DRef<Prims>& W, const RealType* a, const RealType* m, int i, int j, int k) {

    // Scaled 0th order rate 

    double R = nu_i(mol, N_s) * std::exp(-mol.T_bind/T(i,j));

    double mass_per_layer = n_ice_max(N_s, W, a, m, i,j,k) * mol.m_mol;
    double num_layers = ice_grain(i,j,k) / max(mass_per_layer,1e-100);

    ChemRate Rd;
    Rd.rate  = R / (1+num_layers);
    Rd.jac = -R * num_layers / ((1+num_layers)*(1+num_layers));

    return Rd;
}

__host__ __device__
ChemRate R_a_jac(MoleculeRef mol, FieldConstRef<double> T, Field3DRef<Prims>& W, Field3DRef<Ice>& ice, const RealType* m, int i, int j, int k) {
    
    double v_th = std::sqrt(8.*k_B*T(i,j)/(M_PI*mol.m_mol));

    // double R = 3./4. * v_th * W(i,j,k).rho / (4*a[k]*rho_s);
    double R = M_PI * ice(i,j,k).a * ice(i,j,k).a * v_th * W(i,j,k).rho / m[k];

    ChemRate Ra;
    Ra.rate = R;
    Ra.jac = 0.;

    return Ra;
}

__global__ void update_sizegrid(GridRef g, Field3DRef<Ice> ice, Field3DRef<Prims> W, Field3DRef<double> rho_ice, const RealType* m, RealType rho_ms, RealType rho_mi) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {
            for (int k=kidx; k<W.Nd; k+=kstride) {
                double rho_1 = (rho_ice(i,j,k)/(W(i,j,k).rho * rho_mi) + 1./rho_ms);
                ice(i,j,k).a = pow((3.*m[k]/(4.*M_PI)) * rho_1, 1./3.);
                ice(i,j,k).rho = (rho_ice(i,j,k) + W(i,j,k).rho) / (W(i,j,k).rho * rho_1);
            } 
        }
    }

}


__global__ void _implicit_update(GridRef g, Field3DRef<Prims> W, FieldConstRef<double> T, Field3DRef<Ice> ice, const RealType* a, const RealType* m, double N_s, 
                                    MoleculeRef mol, Field3DRef<double> rhos,  Field3DRef<double> rho_ice_0, double dt) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {

            double A = 0, B = 0;

            for (int k=0; k < rho_ice_0.Nd; k++) {
                
                ChemRate R_a = R_a_jac(mol, T, W, ice, m, i,j,k);
                ChemRate R_d = R_d_jac(mol, rhos, N_s, T, W, a, m, i,j,k);

                A += R_d.rate * dt * rho_ice_0(i,j,k) / (1. + R_d.rate * dt);
                B += R_a.rate * dt / (1. + R_d.rate * dt);
            }

            rhos(i,j,rho_ice_0.Nd) = (mol.rho(i,j).vap + A) / (1. + B);

            for (int k=0; k < rho_ice_0.Nd; k++) {

                ChemRate R_a = R_a_jac(mol, T, W, ice, m, i,j,k);
                ChemRate R_d = R_d_jac(mol, rhos, N_s, T, W, a, m, i,j,k);

                rhos(i,j,k) = (rho_ice_0(i,j,k) + R_a.rate * dt * rhos(i,j,rho_ice_0.Nd))  / (1. + R_d.rate * dt);
            }

        }
    }

}

__global__ void copy_initial_values(GridRef g, Field3DRef<double> rhos, MoleculeRef mol, Field3DRef<double> rho_ice, Field3DRef<Prims> w_nof, Field3DRef<Prims> w, FieldRef<Prims> wg, double floor) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    int n_grains = rho_ice.Nd;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {
            
            rhos(i,j,n_grains) = mol.rho(i,j).vap;
            for (int k=0; k<n_grains; k++) {
                rhos(i,j,k) = rho_ice(i,j,k); 
                w_nof(i,j,k).rho = max(w(i,j,k).rho - 1.1*floor*wg(i,j).rho, 0.);
            }
        }
    }

}
__global__ void _copy_rhos(GridRef g, Field3DRef<double> rhos, Field3DRef<double> rhos_0) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {
            for (int k=kidx; k<rhos.Nd+1; k+=kstride) {
            
                rhos_0(i,j,k) = rhos(i,j,k);
            }
        }
    }

}

__global__ void copy_final_values(GridRef g, Field3DRef<double> rhos, MoleculeRef mol, Field3DRef<double> rho_ice, double floor, FieldRef<Prims> wg) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;

    int n_grains = rho_ice.Nd;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {
            
            mol.rho(i,j).vap = rhos(i,j,n_grains)+floor*1e-100*wg(i,j).rho;
            // mol.rho(i,j).ice = mol.rho(i,j).tot - mol.rho(i,j).vap;
            mol.rho(i,j).ice = 0;
            for (int k=0; k<n_grains; k++) {
                rho_ice(i,j,k) = rhos(i,j,k)+floor*1e-100*wg(i,j).rho;
                mol.rho(i,j).ice += rho_ice(i,j,k);
            }
            mol.rho(i,j).tot = mol.rho(i,j).vap + mol.rho(i,j).ice;
        }
    }

}

__global__ void get_tol(Field3DRef<double> rhos, Field3DRef<double> rhos_0, GridRef g, int ngrains, double* err) {

    int iidx = threadIdx.x + blockIdx.x*blockDim.x ;
    int jidx = threadIdx.y + blockIdx.y*blockDim.y ;
    int kidx = threadIdx.z + blockIdx.z*blockDim.z ;
    int istride = gridDim.x * blockDim.x ;
    int jstride = gridDim.y * blockDim.y ;
    int kstride = gridDim.z * blockDim.z ;

    for (int i=iidx+g.Nghost; i<g.NR+g.Nghost; i+=istride) {
        for (int j=jidx+g.Nghost; j<g.Nphi+g.Nghost; j+=jstride) {
            for (int k=kidx; k<ngrains+1; k+=kstride) {
                atomicAdd(&err[0], abs(rhos(i,j,k) - rhos_0(i,j,k)) / (rhos_0(i,j,k) + 1e-100) / ((ngrains + 1) * g.NR * g.Nphi));
            }
        }
    }
}

void IceVapChem::imp_update(double dt) {

    dim3 threads(32,16,1) ;
    dim3 blocks((_g.NR + 2*_g.Nghost+31)/32,(_g.Nphi + 2*_g.Nghost+15)/16,1) ;
          
    dim3 threads2(16,16,4) ;
    dim3 blocks2((_g.NR + 2*_g.Nghost+15)/16,(_g.Nphi + 2*_g.Nghost+15)/16,(_W.Nd+1 + 3)/4) ;

    dim3 threads3(16,16,4) ;
    dim3 blocks3((_g.NR + 2*_g.Nghost+15)/16,(_g.Nphi + 2*_g.Nghost+15)/16, (_W.Nd + 3)/4);
          
    Field3D<double> rhos = Field3D<double>(_g.NR + 2*_g.Nghost, _g.Nphi + 2*_g.Nghost, _W.Nd+1);
    Field3D<double> rhos_0 = Field3D<double>(_g.NR + 2*_g.Nghost, _g.Nphi + 2*_g.Nghost, _W.Nd+1);
    Field3D<Prims> W_nofloor = Field3D<Prims>(_g.NR + 2*_g.Nghost, _g.Nphi + 2*_g.Nghost, _W.Nd);

    copy_initial_values<<<blocks, threads>>>(_g, rhos, _mol, _ice_grains, W_nofloor, _W, _Wg, _floor) ; 

    int it = 0;
    CudaArray<double> err = make_CudaArray<double>(1);
    err[0]= 1;

    while (err[0] > 1e-3) {
        err[0] = 0;

        _copy_rhos<<<blocks2,threads2>>>(_g, rhos, rhos_0);

        _implicit_update<<<blocks,threads>>>(_g, W_nofloor, _T, _sizes.ice, _sizes.grain_sizes(), _sizes.grain_masses(), N_s, _mol, rhos, _ice_grains, dt);

        get_tol<<<blocks2,threads2>>>(rhos, rhos_0, _g, _W.Nd, err.get());

        it++;
        update_sizegrid<<<blocks3,threads3>>>(_g, _sizes.ice, _W, rhos, _sizes.grain_masses(), _sizes.solid_density(), _sizes.ice_density());
        hipDeviceSynchronize();
    }
    // std::cout << "Ice-vap iterations: " << it << "\n";

    copy_final_values<<<blocks,threads>>>(_g, rhos, _mol, _ice_grains, _floor, _Wg);
}


